
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <unistd.h>

int SIZE, ITERATIONS, ANIMATE, BLOCKS, THREADS, SEED, SHARED_MEMORY;
__global__ void play_with_shared_memory(int *in, int *out, int size)
{
    int bid = blockIdx.x;
	int tid = threadIdx.x;
	int bdim = blockDim.x;
	int live_cells = 0;
	int max = size * size;
	int my_id = bid * bdim + tid;
	int mod = my_id % size;
	extern __shared__ int local_board[];

	local_board[tid + size] = in[my_id];
	// Grab neighbors from next block if possible
	if (my_id % bdim >= bdim - size && my_id + size < max)
	{
		local_board[tid + 2 * size] = in[my_id + size];
	}
	// Grab neighbors from previous block if possible
	if (my_id % bdim < size && my_id - size >= 0)
	{
		local_board[tid] = in[my_id - size];
	}
	// Local Id
	int lid = tid + size;
	__syncthreads();

	// Check to see if the index is correct
	if (mod != 0 && my_id + size < max && local_board[lid + size - 1])						// Top left
	{
		live_cells++;
	}
	if (my_id + size < max && local_board[lid + size])													// Top
	{
		live_cells++;
	}
	if (mod != size - 1 && my_id + size < max && local_board[lid + size + 1])				// Top right
	{
		live_cells++;	
	}
	if (mod != 0 && local_board[lid - 1])													// Left
	{
		live_cells++;	
	}
	if (mod != size - 1 && local_board[lid + 1])											// Right
	{
		live_cells++;	
	}
	if (my_id - size>= 0 && mod != 0 && local_board[lid - size - 1])						// Bottom left
	{
		live_cells++;	
	}
	if (my_id - size >= 0 && local_board[lid - size])										// Bottom
	{
		live_cells++;	
	}
	if (my_id - size >= 0 && mod != size - 1 && local_board[lid - size + 1])				// Bottom right
	{
		live_cells++;	
	}

	int is_live = local_board[lid];
	int result = is_live;
	if ((is_live && live_cells < 2) || (is_live && live_cells > 3))
	{
		result = 0;
	}
	else if (!is_live && live_cells == 3)
	{
		result = 1;	
	}
	out[my_id] = result;

	__syncthreads();
}

__global__ void play_with_row_based_index(int *in, int *out, int size)
{
    int bid = blockIdx.x;
	int tid = threadIdx.x;
	int bdim = blockDim.x;
	int live_cells = 0;
	int max = size * size;
	int my_id = bid * bdim + tid;
	int mod = my_id % size;
	// Check to see if the index is correct
	if (mod != 0 && my_id + size < max && in[my_id + size - 1])						// Top left
	{
		live_cells++;
	}
	if (my_id + size < max && in[my_id + size])										// Top
	{
		live_cells++;
	}
	if (mod != size - 1 && my_id + size < max && in[my_id + size + 1])				// Top right
	{
		live_cells++;	
	}
	if (mod != 0 && in[my_id - 1])													// Left
	{
		live_cells++;	
	}
	if (mod != size - 1 && in[my_id + 1])											// Right
	{
		live_cells++;	
	}
	if (my_id - size>= 0 && mod != 0 && in[my_id - size - 1])						// Bottom left
	{
		live_cells++;	
	}
	if (my_id - size >= 0 && in[my_id - size])										// Bottom
	{
		live_cells++;	
	}
	if (my_id - size >= 0 && mod != size - 1 && in[my_id - size + 1])				// Bottom right
	{
		live_cells++;	
	}

	int is_live = in[my_id];
	out[my_id] = is_live;
	if ((is_live && live_cells < 2) || (is_live && live_cells > 3))
	{
		out[my_id] = 0;
	}
	else if (!is_live && live_cells == 3)
	{
		out[my_id] = 1;	
	}
	__syncthreads();
}

__global__ void play(int *in, int *out)
{
    int bid = blockIdx.x;
	int tid = threadIdx.x;
	int bdim = blockDim.x;
	int gdim = gridDim.x;
	int live_cells = 0;
    if (bid * bdim + tid < bdim * gdim)
    {
		// Check to see if the index is correct
		if (bid != 0 && tid != 0 && in[(bid - 1) * bdim + (tid - 1)])
				live_cells++; //Top left
		if (bid != 0 && in[(bid - 1) * bdim + tid])
				live_cells++; //Top	
		if (bid != 0 && tid != bdim - 1 && in[(bid - 1) * bdim + (tid + 1)])
				live_cells++; //Top right
		if (tid != 0 && in[(bid) * bdim + (tid - 1)])
				live_cells++; //left
		//Skipping itself
		if (tid != bdim - 1 && in[(bid) * bdim + (tid + 1)])
				live_cells++; //Right
		if (bid != gdim - 1 && tid != 0 && in[(bid + 1) * bdim + (tid - 1)])
				live_cells++; //Bottom left
		if (bid != gdim - 1 && in[(bid + 1) * bdim + tid])
				live_cells++; //Bottom
		if (bid != gdim - 1 && tid != bdim - 1 && in[(bid + 1) * bdim + (tid + 1)])
				live_cells++; //Bottom right

		int is_live = in[bid * bdim + tid];
		out[bid * bdim + tid] = is_live;
		if ((is_live && live_cells < 2) || (is_live && live_cells > 3))
		{
			out[bid * bdim + tid] = 0;
		}
		else if (!is_live && live_cells == 3)
		{
			out[bid * bdim + tid] = 1;	
		}
    }
	__syncthreads();
}

void print_board(int board[], int size, int iteration)
{
	if (iteration != -1)
	{
		printf("Iteration %d\n", iteration);
	}
    for (int i = 0;i < size; i++)
    {
		for (int j = 0; j < size; j++)
		{
			if (board[i * size + j] != 0 && board[i * size + j] != 1)
			{
				printf("?");	
			}
			else
			{
				if (board[i * size + j])
				{
					printf("\u25A3 ");		
				}
				else
				{
					printf("\u25A2 ");
				}
			}
		}
		printf("\n");
    }
	printf("\n\n");
}

void arg_parse(int argc, char *argv[])
{
	int i = 1;
	char c;
	while(i < argc)
	{
		sscanf(argv[i++], "%c", &c);
		if (c == 's')
		{
			sscanf(argv[i++], "%d", &SIZE);
		}
		if (c == 'a')
		{
			ANIMATE = 1;	
			printf("fu");
		}
		if (c == 'i')
		{
			sscanf(argv[i++], "%d", &ITERATIONS);
		}
		if (c == 'b')
		{
			sscanf(argv[i++], "%d", &BLOCKS);
		}
		if (c == 't')
		{
			sscanf(argv[i++], "%d", &THREADS);
		}
		if (c == 'e')
		{
			sscanf(argv[i++], "%d", &SEED);
		}
		if (c == 'h')
		{
			SHARED_MEMORY = 1;
		}
	}
}

int run()
{
	int animate = ANIMATE != -1 ? ANIMATE : false;
	int size = SIZE ? SIZE : 32;
	int iterations = ITERATIONS ? ITERATIONS : 30;
	int no_blocks = BLOCKS ? BLOCKS : size;
	int no_threads = THREADS ? THREADS : size;
	srand(SEED != -1 ? SEED : time(NULL));
	int *input = (int*)calloc(size * size, sizeof(int));
	int *output = (int*)calloc(size * size, sizeof(int));
	/*int input[16] = {	0, 0, 0, 0, 
						1, 1, 1, 1, 
						0, 0, 0, 0, 
						0, 0, 0, 0};*/
    int *devin, *devout, *devtemp;

    hipMalloc((void**)&devin, size * size * sizeof(int));
    hipMalloc((void**)&devout, size * size * sizeof(int));
	hipMalloc((void**)&devtemp, size * size * sizeof(int));

    for (int i = 0;i < size; i++)
    {
		for (int j = 0; j < size; j++)
		{
			input[i*size + j] = rand() % 2;
		}
    }

	print_board(input, size, 0);

    hipMemcpy(devin, input, size * size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devout, output, size * size * sizeof(int), hipMemcpyHostToDevice);

	int shared_board_size = (no_threads + 2 * size) * sizeof(int);
	// Call the kernel for one iteration
	clock_t start = clock(), diff;
	for (int i = 0;i<iterations;i++)
	{
		if (i == 0)
		{
			//play_with_row_based_index<<<no_blocks,no_threads,shared_board_size>>>(devin, devout, size);
			play_with_shared_memory<<<no_blocks,no_threads,shared_board_size>>>(devin, devout, size);
		}
		else
		{
			//play_with_row_based_index<<<no_blocks,no_threads,shared_board_size>>>(devtemp, devout, size);
			play_with_shared_memory<<<no_blocks,no_threads,shared_board_size>>>(devtemp, devout, size);
		}
		hipMemcpy(devtemp, devout, size * size * sizeof(int), hipMemcpyDeviceToDevice);
		hipMemcpy(output, devout, size * size * sizeof(int), hipMemcpyDeviceToHost);
		if (animate == true)
		{
			system("clear");
			print_board(output, size, i);
			usleep(100000);
		}
	}

	// Copy back the output
    hipMemcpy(output, devout, size * size * sizeof(int), hipMemcpyDeviceToHost);
	
	diff = clock() - start;
	int msec = diff * 1000 / CLOCKS_PER_SEC;
	printf("Time in kernel: %d seconds %d milliseconds\n", msec / 1000, msec % 1000);

	print_board(output, size, iterations);

	// Free device memory
    hipFree(devin);
    hipFree(devout);
	hipFree(devtemp);

    return 0;
}

int main(int argc, char* argv[])
{
	SIZE = 0, ITERATIONS = 0, ANIMATE = -1, BLOCKS = 0, THREADS = 0, SHARED_MEMORY = 0, SEED = -1;
	arg_parse(argc, argv);
	run();
	return 0;
}
